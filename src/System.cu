#include "hip/hip_runtime.h"

#include "EnsemblePreprocessor.h"
#include "System.h"

#include "CalculateEnergy.h"
#include "EnergyTypes.h"
#include "Setup.h"               //For source of setup data.
#include "ConfigSetup.h"         //For types directly read from config. file
#include "StaticVals.h"
#include "Molecules.h"           //For indexing molecules.
#include "MoveConst.h"           //For array of move objects.
#include "MoveBase.h"            //For move bases....
#include "MoleculeTransfer.h"


#include <hip/hip_runtime.h> 

System::System(StaticVals& statics) : 
statV(statics),
#ifdef VARIABLE_VOLUME
	boxDimRef(boxDimensions),
#else
	boxDimRef(statics.boxDimensions),
#endif
#ifdef VARIABLE_PARTICLE_NUMBER
	molLookupRef(molLookup),
#else
	molLookupRef(statics.molLookup),
#endif
	prng(molLookupRef),
	coordinates(boxDimRef, com, molLookupRef, prng, statics.mol),
	com(boxDimRef, coordinates, molLookupRef, statics.mol),
	moveSettings(boxDimRef),
	calcEnergy(statics, *this) {}


void System::LoadDataToGPU()

{
	uint numKinds = molLookupRef.GetNumKind();
	uint * numByBox = new uint [BOX_TOTAL];


	uint * numByKindBox = new uint [BOX_TOTAL * numKinds];
	uint * numAtomsByBox = new uint[BOX_TOTAL];
	molLookupRef.TotalAtomsMols(numByBox, numByKindBox, numAtomsByBox, calcEnergy.mols.kinds);

	for (int i=0;i<BOX_TOTAL;i++)
	{calcEnergy.AtomCount[i] = numAtomsByBox[i];

	}

	if (BOX_TOTAL==1)
		calcEnergy.MolCount[0] = calcEnergy.mols.count;

	else
		for (int i=0;i<BOX_TOTAL;i++)
		{
			calcEnergy.MolCount[i] = molLookupRef.NumInBox(i);

		}


		hipMalloc((void**)&calcEnergy.Gpu_Potential, sizeof(SystemPotential));

		int count2 = calcEnergy.forcefield.particles->NumKinds() * calcEnergy.forcefield.particles->NumKinds();
		hipMalloc((void**)&calcEnergy.Gpu_sigmaSq , sizeof(double) * count2);
		hipMalloc((void**)&calcEnergy.Gpu_epsilon_cn , sizeof(double) * count2);
		hipMalloc((void**)&calcEnergy.Gpu_epsilon_cn_6 , sizeof(double) * count2);
		hipMalloc((void**)&calcEnergy.Gpu_nOver6 , sizeof(double) * count2);
		hipMalloc((void**)&calcEnergy.Gpu_enCorrection , sizeof(double) * count2);
		hipMalloc((void**)&calcEnergy.Gpu_virCorrection , sizeof(double) * count2);
		hipMemcpy(calcEnergy.Gpu_sigmaSq, calcEnergy.forcefield.particles->sigmaSq, sizeof(double)*count2, hipMemcpyHostToDevice);
		hipMemcpy(calcEnergy.Gpu_epsilon_cn, calcEnergy.forcefield.particles->epsilon_cn, sizeof(double)*count2, hipMemcpyHostToDevice);
		hipMemcpy(calcEnergy.Gpu_epsilon_cn_6, calcEnergy.forcefield.particles->epsilon_cn_6, sizeof(double)*count2, hipMemcpyHostToDevice);
		hipMemcpy(calcEnergy.Gpu_nOver6, calcEnergy.forcefield.particles->nOver6, sizeof(double)*count2, hipMemcpyHostToDevice);
		hipMemcpy(calcEnergy.Gpu_enCorrection, calcEnergy.forcefield.particles->enCorrection, sizeof(double)* count2, hipMemcpyHostToDevice);
		hipMemcpy(calcEnergy.Gpu_virCorrection, calcEnergy.forcefield.particles->virCorrection, sizeof(double)*count2, hipMemcpyHostToDevice);


		hipMalloc((void**) &calcEnergy.dev_EnergyContrib,  BLOCK_SIZE * sizeof(double));
		hipMalloc((void**) &calcEnergy.dev_VirialContrib,  BLOCK_SIZE * sizeof(double));


#ifdef MIE_INT_ONLY
		hipMalloc ( (void**)  &calcEnergy.Gpu_partn , sizeof(uint) * count2);
		hipMemcpy(calcEnergy.Gpu_partn,    calcEnergy.forcefield.particles.n,  sizeof(uint) * count2  ,    hipMemcpyHostToDevice);
#else
		hipMalloc ( (void**)  &calcEnergy.Gpu_partn , sizeof(double) * count2);
		hipMemcpy(calcEnergy.Gpu_partn,    calcEnergy.forcefield.particles->n,  sizeof(double) * count2  ,  hipMemcpyHostToDevice);
#endif


		hipMalloc ( (void**)  & calcEnergy.Gpu_x, sizeof(double) * calcEnergy.currentCoords.Count());
		hipMalloc ( (void**)  & calcEnergy.Gpu_y, sizeof(double) * calcEnergy.currentCoords.Count());
		hipMalloc ( (void**)  & calcEnergy.Gpu_z, sizeof(double) * calcEnergy.currentCoords.Count());
		hipMemcpy(calcEnergy.Gpu_x, calcEnergy.currentCoords.x, sizeof(double) *calcEnergy.currentCoords.Count() ,  hipMemcpyHostToDevice);
		hipMemcpy(calcEnergy.Gpu_y, calcEnergy.currentCoords.y, sizeof(double) *calcEnergy.currentCoords.Count() ,  hipMemcpyHostToDevice);
		hipMemcpy(calcEnergy.Gpu_z, calcEnergy.currentCoords.z, sizeof(double) *calcEnergy.currentCoords.Count() ,  hipMemcpyHostToDevice);

		calcEnergy.cordsx= (double *)malloc (sizeof(double) * calcEnergy.currentCoords.Count());
		calcEnergy.cordsy= (double *)malloc (sizeof(double) * calcEnergy.currentCoords.Count());
		calcEnergy.cordsz= (double *)malloc (sizeof(double) * calcEnergy.currentCoords.Count());

#if ENSEMBLE == GEMC || ENSEMBLE == GCMC
		// for mol transfer
		calcEnergy.tmpx = (double*) malloc (sizeof(double) * calcEnergy.currentCoords.Count());
		calcEnergy.tmpy = (double*) malloc (sizeof(double) * calcEnergy.currentCoords.Count());
		calcEnergy.tmpz = (double*) malloc (sizeof(double) * calcEnergy.currentCoords.Count());


		calcEnergy.tmpCOMx = (double*) malloc (sizeof(double) * com.Count());
		calcEnergy.tmpCOMy = (double*) malloc  (sizeof(double) * com.Count());
		calcEnergy.tmpCOMz = (double*) malloc ( sizeof(double) * com.Count());
		calcEnergy.tmpMolStart = (uint*) malloc ( sizeof(uint) * (calcEnergy.mols.count + 1));
#endif


		// mols data
		hipMalloc ( (void**)  &calcEnergy.Gpu_start , sizeof(uint) * (calcEnergy.mols.count + 1));
		hipMalloc ( (void**)  &calcEnergy.Gpu_kIndex , sizeof(uint) * calcEnergy.mols.resKindsCount);
		hipMalloc ( (void**)  &calcEnergy.Gpu_countByKind , sizeof(uint) * calcEnergy.mols.kindsCount);
		hipMalloc ( (void**)  &calcEnergy.Gpu_pairEnCorrections , sizeof(double) * calcEnergy.mols.kindsCount * calcEnergy.mols.kindsCount );
		hipMalloc ( (void**)  &calcEnergy.Gpu_pairVirCorrections , sizeof(double) * calcEnergy.mols.kindsCount * calcEnergy.mols.kindsCount );
		hipMemcpy(calcEnergy.Gpu_start,    calcEnergy.mols.start,  sizeof(uint) * (calcEnergy.mols.count + 1) ,   hipMemcpyHostToDevice);
		hipMemcpy(calcEnergy.Gpu_kIndex,   calcEnergy.mols.kIndex, sizeof(uint) * (calcEnergy.mols.resKindsCount) , hipMemcpyHostToDevice);
		hipMemcpy(calcEnergy.Gpu_countByKind,  calcEnergy.mols.countByKind,    sizeof(uint) * (calcEnergy.mols.kindsCount) ,    hipMemcpyHostToDevice);
		hipMemcpy(calcEnergy.Gpu_pairEnCorrections,    calcEnergy.mols.pairEnCorrections,  sizeof(double) * (calcEnergy.mols.kindsCount * calcEnergy.mols.kindsCount) , hipMemcpyHostToDevice);
		hipMemcpy(calcEnergy.Gpu_pairVirCorrections,   calcEnergy.mols.pairVirCorrections, sizeof(double) * (calcEnergy.mols.kindsCount * calcEnergy.mols.kindsCount) , hipMemcpyHostToDevice);
		calcEnergy.CPU_atomKinds = (uint *) malloc (sizeof(uint) * (calcEnergy.currentCoords.Count()) );
		calcEnergy.atmsPerMol = (uint *) malloc (sizeof(uint) * (calcEnergy.mols.count) );


		hipMalloc ( (void**)  &calcEnergy.atomsMoleculeNo , sizeof(uint) * (calcEnergy.currentCoords.Count()));

		calcEnergy.CPU_atomsMoleculeNo =  (uint *) malloc (sizeof(uint) * (calcEnergy.currentCoords.Count()) );

		int ctr = 0;
		uint MaxLen=0;

		for (int i = 0; i < calcEnergy.mols.count; i++) {
			calcEnergy.atmsPerMol[i] = calcEnergy.mols.kinds[calcEnergy.mols.kIndex[i]].numAtoms;


			if (calcEnergy.mols.kinds[calcEnergy.mols.kIndex[i]].numAtoms > MaxLen)
				MaxLen= calcEnergy.mols.kinds[calcEnergy.mols.kIndex[i]].numAtoms;


			for (int j = 0; j < calcEnergy.mols.kinds[calcEnergy.mols.kIndex[i]].numAtoms; j++ ) {

				calcEnergy.CPU_atomKinds[ctr] = calcEnergy.mols.kinds[calcEnergy.mols.kIndex[i]].atomKind[j];
				calcEnergy.CPU_atomsMoleculeNo[ctr] = i;

				ctr++;
			}
		}



		hipMalloc((void**) &calcEnergy.tempCoordsX,  2*MaxLen * sizeof(double));

		hipMalloc((void**) &calcEnergy.tempCoordsY,  2*MaxLen * sizeof(double));

		hipMalloc((void**) &calcEnergy.tempCoordsZ,  2*MaxLen * sizeof(double));
		hipMalloc((void**)&calcEnergy.dev_partEnergy, MaxTrialNumber * sizeof(double));


		hipMalloc((void**) &calcEnergy.trialPosX,  MaxTrialNumber * sizeof(double));

		hipMalloc((void**) &calcEnergy.trialPosY,  MaxTrialNumber * sizeof(double));

		hipMalloc((void**) &calcEnergy.trialPosZ, MaxTrialNumber* sizeof(double));

		calcEnergy.MaxTrialNumber=MaxTrialNumber;

		calcEnergy.FinalEnergyNVirial= (double*) malloc (sizeof(double) * MaxTrialNumber);



		hipMalloc ( (void**)  &calcEnergy.Gpu_atomKinds , sizeof(uint) * (calcEnergy.currentCoords.Count()));
		hipMemcpy(calcEnergy.Gpu_atomKinds, calcEnergy.CPU_atomKinds, sizeof(uint) * (calcEnergy.currentCoords.Count()), hipMemcpyHostToDevice );
		hipMalloc ( (void**)  &calcEnergy.NoOfAtomsPerMol , sizeof(uint) * (calcEnergy.mols.count));
		hipMemcpy(calcEnergy.NoOfAtomsPerMol, calcEnergy.atmsPerMol, sizeof(uint) * (calcEnergy.mols.count), hipMemcpyHostToDevice );



		hipMalloc ( (void**)  &calcEnergy.Gpu_COMX , sizeof(double) * com.Count());
		hipMalloc ( (void**)  &calcEnergy.Gpu_COMY , sizeof(double) * com.Count());
		hipMalloc ( (void**)  &calcEnergy.Gpu_COMZ , sizeof(double) * com.Count());
		hipMemcpy(calcEnergy.Gpu_COMX, com.x, sizeof(double) *  com.Count(), hipMemcpyHostToDevice );
		hipMemcpy(calcEnergy.Gpu_COMY, com.y, sizeof(double) *  com.Count() , hipMemcpyHostToDevice);
		hipMemcpy(calcEnergy.Gpu_COMZ, com.z, sizeof(double) *  com.Count(), hipMemcpyHostToDevice );


		hipMemcpy(calcEnergy.atomsMoleculeNo,calcEnergy.CPU_atomsMoleculeNo , sizeof(uint) * (calcEnergy.currentCoords.Count()), hipMemcpyHostToDevice );



		hipMalloc ( (void **) &calcEnergy.Gpu_result, sizeof(bool) );

#if ENSEMBLE == GEMC
		hipMalloc ( (void**)  &calcEnergy.newCOMX , sizeof(double) * com.Count());
		hipMalloc ( (void**)  &calcEnergy.newCOMY , sizeof(double) * com.Count());
		hipMalloc ( (void**)  &calcEnergy.newCOMZ , sizeof(double) * com.Count());
		hipMemcpy(calcEnergy.newCOMX, calcEnergy.Gpu_COMX, sizeof(double) *  com.Count() , hipMemcpyDeviceToDevice);
		hipMemcpy(calcEnergy.newCOMY, calcEnergy.Gpu_COMY, sizeof(double) *  com.Count() , hipMemcpyDeviceToDevice);
		hipMemcpy(calcEnergy.newCOMZ, calcEnergy.Gpu_COMZ, sizeof(double) *  com.Count() , hipMemcpyDeviceToDevice);
		hipMalloc ( (void**)  & calcEnergy.newX, sizeof(double) * calcEnergy.currentCoords.Count());
		hipMalloc ( (void**)  & calcEnergy.newY, sizeof(double) * calcEnergy.currentCoords.Count());
		hipMalloc ( (void**)  & calcEnergy.newZ, sizeof(double) * calcEnergy.currentCoords.Count());
#endif


		// streams
		hipStreamCreate(&calcEnergy.stream0);
		hipStreamCreate(&calcEnergy.stream1);


		printf("Data load to GPU done!\n");
		hipDeviceSynchronize();
		hipError_t  code = hipGetLastError();

		if (code != hipSuccess) {
			printf ("Cuda error at end of data load to GPU -- %s\n", hipGetErrorString(code));
			exit(2);
		}



}




__global__ void InitCells(
	double *Gpu_x,
	double *Gpu_y,
	double *Gpu_z,
	unsigned int * atomCells,
	unsigned int * atomCountrs,
	int boxOffset, 
	int cellsxDim,
	int cellsyDim,
	int cellszDim,
	int NumberOfCellsInBox,
	int cellOffset,
	int atomCount
	)
{

	int threadId = blockIdx.x * blockDim.x + threadIdx.x ;
	if (threadId< atomCount)
	{
		int atomId = threadId + boxOffset ;// offset for box atoms 

		int xCellPos= ((int)Gpu_x[atomId]>>HALF_MICROCELL_DIM);
		int yCellPos= ((int)Gpu_y[atomId]>>HALF_MICROCELL_DIM);
		int zCellPos= ((int)Gpu_z[atomId]>>HALF_MICROCELL_DIM);

		int position = (zCellPos * cellszDim + yCellPos)* cellsyDim +xCellPos;// flat 3d to 1d 

		int OldValue = atomicInc(&(atomCountrs[position+cellOffset]), MAX_ATOMS_PER_CELL);


		if (OldValue < MAX_ATOMS_PER_CELL)
			atomCells[(OldValue * NumberOfCellsInBox + position)+ cellOffset*MAX_ATOMS_PER_CELL ] = atomId;
		else {
			printf("System is trying to initialize the system with too many particles in Cell %d.\n", position);

		}


	}

}


__device__ int GetMinimumCells(int CellCoord, const int CellsPerDimension) {
	if (CellCoord >= CellsPerDimension)
		CellCoord -= CellsPerDimension;
	else if (CellCoord < 0)
		CellCoord += CellsPerDimension;
	return CellCoord;
}




void System::LoadMolsToCells()
{

	int TotalCellsInSystem=0;

	for (int i=0; i < BOX_TOTAL; i++)
	{


		calcEnergy.CellsPerDim[i*3] =    (int) ceil(boxDimRef.axis.x[i]/MICROCELL_DIM);
		calcEnergy.CellsPerDim[i*3+1] = (int) ceil(boxDimRef.axis.y[i]/MICROCELL_DIM);
		calcEnergy.CellsPerDim[i*3+2] = (int) ceil(boxDimRef.axis.z[i]/MICROCELL_DIM);

		calcEnergy.TotalCellsPerBox[i] = calcEnergy.CellsPerDim[i*3]* calcEnergy.CellsPerDim[i*3+1]*calcEnergy.CellsPerDim[i*3+2];


		calcEnergy.EdgeAdjust[i*3]= calcEnergy.CellsPerDim[i*3]*MICROCELL_DIM - boxDimRef.axis.x[i];
		calcEnergy.EdgeAdjust[i*3+1]= calcEnergy.CellsPerDim[i*3+1] *MICROCELL_DIM- boxDimRef.axis.y[i];
		calcEnergy.EdgeAdjust[i*3+2]= calcEnergy.CellsPerDim[i*3+2]*MICROCELL_DIM - boxDimRef.axis.z[i];

		calcEnergy.CellDim[i*3] =  min(calcEnergy.CellsPerDim[i*3], int(2.0 * calcEnergy.currentAxes.rCut + calcEnergy.EdgeAdjust[i*3])/MICROCELL_DIM + 2);
		calcEnergy.CellDim[i*3+1] =  min(calcEnergy.CellsPerDim[i*3+1], int(2.0 * calcEnergy.currentAxes.rCut + calcEnergy.EdgeAdjust[i*3+1])/MICROCELL_DIM + 2);
		calcEnergy.CellDim[i*3+2] =  min(calcEnergy.CellsPerDim[i*3+2], int(2.0 * calcEnergy.currentAxes.rCut + calcEnergy.EdgeAdjust[i*3+2])/MICROCELL_DIM + 2);

		TotalCellsInSystem+=calcEnergy.TotalCellsPerBox[i]; 
	}




	calcEnergy.BlockSize.x = BLOCK_DIM;
	calcEnergy.BlockSize.y = calcEnergy.BlockSize.x;
	calcEnergy.BlockSize.z = calcEnergy.BlockSize.x;

	hipMalloc((void **) &calcEnergy.atomCountrs, TotalCellsInSystem * sizeof(uint));
	hipMemset((void *)  calcEnergy.atomCountrs, 0, TotalCellsInSystem * sizeof(uint));
	hipMalloc((void **) &calcEnergy.atomCells, MAX_ATOMS_PER_CELL * TotalCellsInSystem * sizeof(uint));



	int ThreadsPerBlock=0;
	int BlocksPerGrid=0;

	for (int i=0; i < BOX_TOTAL; i++)
	{
		if (calcEnergy.AtomCount[i] < MAXTHREADSPERBLOCK)
			ThreadsPerBlock = calcEnergy.AtomCount[i];
		else
			ThreadsPerBlock = MAXTHREADSPERBLOCK;



		BlocksPerGrid = (calcEnergy.AtomCount[i]+ ThreadsPerBlock - 1) / ThreadsPerBlock;

		InitCells <<<BlocksPerGrid,ThreadsPerBlock>>> (
			calcEnergy.Gpu_x,
			calcEnergy.Gpu_y,
			calcEnergy.Gpu_z,
			calcEnergy.atomCells,
			calcEnergy.atomCountrs,
			(i==0)?0:calcEnergy.AtomCount[i-1], 
			calcEnergy.CellsPerDim[i*3],
			calcEnergy.CellsPerDim[i*3+1],
			calcEnergy.CellsPerDim[i*3+2],
			calcEnergy.TotalCellsPerBox[i],
			(i==0)?0:calcEnergy.TotalCellsPerBox[i-1],
			calcEnergy.AtomCount[i]
		); 


	}
	hipDeviceSynchronize();

	hipError_t  code = hipGetLastError();
	if (code != hipSuccess) {
		printf ("Cuda error at Load cells - %s\n", hipGetErrorString(code));
		exit(2);
	}



}


void System::FreeGPUDATA()
{   // free CUDA variables 
	hipFree(calcEnergy.Gpu_Potential);
	hipFree(calcEnergy.Gpu_sigmaSq);
	hipFree(calcEnergy.Gpu_epsilon_cn);
	hipFree(calcEnergy.Gpu_epsilon_cn_6);
	hipFree(calcEnergy.Gpu_nOver6);
	hipFree(calcEnergy.Gpu_enCorrection);
	hipFree(calcEnergy.Gpu_virCorrection);
	hipFree(calcEnergy.Gpu_partn);
	hipFree(calcEnergy.Gpu_x);
	hipFree(calcEnergy.Gpu_y);
	hipFree(calcEnergy.Gpu_z);
	hipFree(calcEnergy.Gpu_start);
	hipFree(calcEnergy.Gpu_kIndex);
	hipFree(calcEnergy.Gpu_countByKind);
	hipFree(calcEnergy.Gpu_pairEnCorrections);
	hipFree(calcEnergy.Gpu_pairVirCorrections);
	hipFree(calcEnergy.Gpu_atomKinds);
	hipFree(calcEnergy.NoOfAtomsPerMol);
	hipFree(calcEnergy.Gpu_COMX);
	hipFree(calcEnergy.Gpu_COMY);
	hipFree(calcEnergy.Gpu_COMZ);
	hipFree(calcEnergy.Gpu_result);
	hipFree(calcEnergy.newX);
	hipFree(calcEnergy.newY);
	hipFree(calcEnergy.newZ);
	hipFree(calcEnergy.newCOMX);
	hipFree(calcEnergy.newCOMY);
	hipFree(calcEnergy.newCOMZ);
	hipFree(calcEnergy.dev_partEnergy);
	hipFree(calcEnergy.trialPosX);

	hipFree(calcEnergy.trialPosY);

	hipFree(calcEnergy.trialPosZ);
	// free streams
	hipStreamDestroy(calcEnergy.stream0);
	hipStreamDestroy(calcEnergy.stream1);

	// free cpu arrays used for copy

	free(calcEnergy.CPU_atomKinds);
	free(calcEnergy.atmsPerMol);
	free(calcEnergy.CPU_atomsMoleculeNo);
	free(calcEnergy.FinalEnergyNVirial);
	free(calcEnergy.cordsx);
	free(calcEnergy.cordsy);
	free(calcEnergy.cordsz);
	#if ENSEMBLE == GEMC || ENSEMBLE == GCMC
	free(calcEnergy.tmpx);
	free(calcEnergy.tmpy);
	free(calcEnergy.tmpz);
	free(calcEnergy.tmpCOMx);
	free(calcEnergy.tmpCOMy);
	free(calcEnergy.tmpCOMz);
	free(calcEnergy.tmpMolStart);

	#endif


}

System::~System()
{  FreeGPUDATA();//  
delete moves[mv::DISPLACE];
delete moves[mv::ROTATE];
#if ENSEMBLE == GEMC
delete moves[mv::VOL_TRANSFER];
#endif
#if ENSEMBLE == GEMC || ENSEMBLE == GCMC
delete moves[mv::MOL_TRANSFER];
#endif
}

// Beginning of GPU Architecture definitions
inline int System::_ConvertSMVer2Cores(int major, int minor)
{
	// Defines for GPU Architecture types (using the SM version to determine the # of cores per SM)
	typedef struct {
		int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] = {
		{ 0x10,  8 }, // Tesla Generation (SM 1.0) G80 class
		{ 0x11,  8 }, // Tesla Generation (SM 1.1) G8x class
		{ 0x12,  8 }, // Tesla Generation (SM 1.2) G9x class
		{ 0x13,  8 }, // Tesla Generation (SM 1.3) GT200 class
		{ 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
		{ 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
		{ 0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
		{ 0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
		{ 0x50, 128}, // Maxwell Generation (SM 5.0) GM10x class
		{   -1, -1 }
	};

	int index = 0;
	while (nGpuArchCoresPerSM[index].SM != -1) {
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor) ) {
			return nGpuArchCoresPerSM[index].Cores;
		}
		index++;
	}
	printf("MapSMtoCores undefined SMversion %d.%d!\n", major, minor);
	return -1;
}

// conv cell list

__global__ void GenerateAdjacencyCellList(const float CellSize,
	const int NumberOfCells, int dev_AdjacencyCellList[], int step, int box) {

		int CellId = threadIdx.x + blockIdx.x * blockDim.x;
		int x, y, z;
		if (CellId < NumberOfCells * NumberOfCells * NumberOfCells) {

			int xCell, yCell, zCell;
			zCell = (int) (CellId / NumberOfCells);
			yCell = zCell % NumberOfCells;
			zCell = (int) (zCell / NumberOfCells);
			xCell = CellId % NumberOfCells;
			int Counter = 0;


			for (int i = -1; i < 2; i++) {
				x = xCell;
				x += i;
				if (x < 0)
					x = NumberOfCells - 1;
				else if (x >= NumberOfCells)
					x = 0;
				for (int j = -1; j < 2; j++) {
					y = yCell;
					y += j;
					if (y < 0)
						y = NumberOfCells - 1;
					else if (y >= NumberOfCells)
						y = 0;
					for (int k = -1; k < 2; k++) {
						z = zCell;
						z += k;
						if (z < 0)
							z = NumberOfCells - 1;
						else if (z >= NumberOfCells)
							z = 0;

						dev_AdjacencyCellList[(Counter) + CellId * 27] = x+ y * NumberOfCells+ z * NumberOfCells * NumberOfCells;

						Counter++;

					} // for k

				} // for j
			} // for i
		} // if this is a valid Cell number
} // end of the function




void  System::CreateAdjCellList()
{


	int ThreadsPerBlock,BlocksPerGrid;

	for (int i=0; i < BOX_TOTAL; i++)
	{

		//Calculate the number of cells in the box from the cell size
		calcEnergy.NumberOfCells[i] = (int) floor(boxDimRef.axis.x[i] / calcEnergy.currentAxes.rCut);

		//Calculate the cell size from the number of cells
		calcEnergy.CellSize[i] = boxDimRef.axis.x[i] / (float) calcEnergy.NumberOfCells[i];

		calcEnergy.TotalNumberOfCells[i]     = calcEnergy.NumberOfCells[i] * calcEnergy.NumberOfCells[i] * calcEnergy.NumberOfCells[i];
		calcEnergy.AdjacencyCellList_size[i] = sizeof(int) * calcEnergy.TotalNumberOfCells[i] * 27;



		if (i==0)
		{
			hipMalloc((void**) &calcEnergy.dev_AdjacencyCellList0, calcEnergy.AdjacencyCellList_size[0]);

		}

#if ENSEMBLE == GEMC
		else
		{
			hipMalloc((void**) &calcEnergy.dev_AdjacencyCellList1, calcEnergy.AdjacencyCellList_size[1]);


		}
#endif

		if (calcEnergy.TotalNumberOfCells[i] < MAXTHREADSPERBLOCK)
			ThreadsPerBlock = calcEnergy.TotalNumberOfCells[i];
		else
			ThreadsPerBlock = MAXTHREADSPERBLOCK;
		BlocksPerGrid = (calcEnergy.TotalNumberOfCells[i] + ThreadsPerBlock - 1)
			/ ThreadsPerBlock;


		if (i==0)
			GenerateAdjacencyCellList<<<BlocksPerGrid, ThreadsPerBlock>>>(calcEnergy.CellSize[i], calcEnergy.NumberOfCells[i], calcEnergy.dev_AdjacencyCellList0,step,0);

#if ENSEMBLE == GEMC
		else
			GenerateAdjacencyCellList<<<BlocksPerGrid, ThreadsPerBlock>>>(calcEnergy.CellSize[i], calcEnergy.NumberOfCells[i], calcEnergy.dev_AdjacencyCellList1,step,1);
#endif

		hipDeviceSynchronize();



	}


	hipMalloc((void**) &calcEnergy.dev_CountAtomsInCell0,
		sizeof(uint) * calcEnergy.TotalNumberOfCells[0]);


	hipMemset(calcEnergy.dev_CountAtomsInCell0, 0, sizeof(uint) * calcEnergy.TotalNumberOfCells[0]);


	hipMalloc((void**) &calcEnergy.AtomsInCells0,
		sizeof(int) * calcEnergy.TotalNumberOfCells[0] * MaxParticleInCell);

	hipMemset(calcEnergy.AtomsInCells0, 0,
		sizeof(int) * calcEnergy.TotalNumberOfCells[0] * MaxParticleInCell);


#if ENSEMBLE == GEMC
	hipMalloc((void**) &calcEnergy.dev_CountAtomsInCell1,
		sizeof(uint) * calcEnergy.TotalNumberOfCells[1]);


	hipMemset(calcEnergy.dev_CountAtomsInCell1, 0, sizeof(uint) * calcEnergy.TotalNumberOfCells[1]);


	hipMalloc((void**) &calcEnergy.AtomsInCells1,
		sizeof(int) * calcEnergy.TotalNumberOfCells[1] * MaxParticleInCell);

	hipMemset(calcEnergy.AtomsInCells1, 0,
		sizeof(int) * calcEnergy.TotalNumberOfCells[1] * MaxParticleInCell);

#endif



}
#if ENSEMBLE == GEMC
void  System::CreateAdjCellListForScaledMols(uint majKind)
{


	int ThreadsPerBlock,BlocksPerGrid;

	for (int i=0; i < BOX_TOTAL; i++)
	{

		//Calculate the number of cells in the box from the cell size
		calcEnergy.NumberOfCells[i] = (int) floor(((VolumeTransfer*)moves[majKind])->newDim.axis.x[i] / calcEnergy.currentAxes.rCut);

		//Calculate the cell size from the number of cells
		calcEnergy.CellSize[i] = ((VolumeTransfer*)moves[majKind])->newDim.axis.x[i] / (double) calcEnergy.NumberOfCells[i];



		calcEnergy.TotalNumberOfCells[i]     = calcEnergy.NumberOfCells[i] * calcEnergy.NumberOfCells[i] * calcEnergy.NumberOfCells[i];
		calcEnergy.AdjacencyCellList_size[i] = sizeof(int) * calcEnergy.TotalNumberOfCells[i] * 27;

		if (i==0)
		{
			hipMalloc((void**) &calcEnergy.dev_AdjacencyCellList0, calcEnergy.AdjacencyCellList_size[0]);

		}
		else
		{
			hipMalloc((void**) &calcEnergy.dev_AdjacencyCellList1, calcEnergy.AdjacencyCellList_size[1]);


		}

		if (calcEnergy.TotalNumberOfCells[i] < MAXTHREADSPERBLOCK)
			ThreadsPerBlock = calcEnergy.TotalNumberOfCells[i];
		else
			ThreadsPerBlock = MAXTHREADSPERBLOCK;
		BlocksPerGrid = (calcEnergy.TotalNumberOfCells[i] + ThreadsPerBlock - 1)
			/ ThreadsPerBlock;






		if (i==0)
		{GenerateAdjacencyCellList<<<BlocksPerGrid, ThreadsPerBlock>>>(calcEnergy.CellSize[i], calcEnergy.NumberOfCells[i], calcEnergy.dev_AdjacencyCellList0,step,0);


		}
		else
			GenerateAdjacencyCellList<<<BlocksPerGrid, ThreadsPerBlock>>>(calcEnergy.CellSize[i], calcEnergy.NumberOfCells[i], calcEnergy.dev_AdjacencyCellList1,step,1);

		hipDeviceSynchronize();



	}

	hipMalloc((void**) &calcEnergy.dev_CountAtomsInCell0,
		sizeof(uint) * calcEnergy.TotalNumberOfCells[0]);


	hipMemset(calcEnergy.dev_CountAtomsInCell0, 0, sizeof(uint) * calcEnergy.TotalNumberOfCells[0]);


	hipMalloc((void**) &calcEnergy.AtomsInCells0,
		sizeof(int) * calcEnergy.TotalNumberOfCells[0] * MaxParticleInCell);

	hipMemset(calcEnergy.AtomsInCells0, 0,
		sizeof(int) * calcEnergy.TotalNumberOfCells[0] * MaxParticleInCell);



	hipMalloc((void**) &calcEnergy.dev_CountAtomsInCell1,
		sizeof(uint) * calcEnergy.TotalNumberOfCells[1]);


	hipMemset(calcEnergy.dev_CountAtomsInCell1, 0, sizeof(uint) * calcEnergy.TotalNumberOfCells[1]);


	hipMalloc((void**) &calcEnergy.AtomsInCells1,
		sizeof(int) * calcEnergy.TotalNumberOfCells[1] * MaxParticleInCell);

	hipMemset(calcEnergy.AtomsInCells1, 0,
		sizeof(int) * calcEnergy.TotalNumberOfCells[1] * MaxParticleInCell);


}
#endif


__global__ void InitializeParticlePositions(
	double *Gpu_x,
	double *Gpu_y,
	double *Gpu_z,
	unsigned int NumberParticlesInBox,
	double BoxLengthXAxis, double BoxLengthYAxis, double BoxLengthZAxis,
	int boxOffset ,
	double CellSize, int NumberOfCells, 
	uint dev_CountAtomsInCell[],
	int AtomsInCells[],
	int step
	) {


		int ParticleNumber = blockIdx.x * blockDim.x + threadIdx.x;


		if (ParticleNumber < NumberParticlesInBox) {



			int xCell = (int) (Gpu_x[ParticleNumber+ boxOffset ] / CellSize);
			int yCell = (int) (Gpu_y[ParticleNumber+ boxOffset ] / CellSize);
			int zCell = (int) (Gpu_z[ParticleNumber+ boxOffset ] / CellSize);
			int CellId = xCell + yCell * NumberOfCells
				+ zCell * NumberOfCells * NumberOfCells;


			int OldValue = atomicInc(&(dev_CountAtomsInCell[CellId]), MaxParticleInCell);



			if (OldValue < MaxParticleInCell)

				AtomsInCells[OldValue+ CellId * MaxParticleInCell] = ParticleNumber;

			else {
				printf("Conv cell System is trying to initialize the system with too many particles in Cell %d.\n", CellId);

			}	

		}
}



void System::LoadAtomsToCells()
{
	int ThreadsPerBlock, BlocksPerGrid;


	if ((calcEnergy.AtomCount[0]) < MAXTHREADSPERBLOCK)
		ThreadsPerBlock = calcEnergy.AtomCount[0];
	else
		ThreadsPerBlock = MAXTHREADSPERBLOCK;

	BlocksPerGrid = (calcEnergy.AtomCount[0] + ThreadsPerBlock - 1)/ ThreadsPerBlock;




	InitializeParticlePositions<<<BlocksPerGrid, ThreadsPerBlock>>>(
		calcEnergy.Gpu_x,
		calcEnergy.Gpu_y,
		calcEnergy.Gpu_z,
		calcEnergy.AtomCount[0],
		boxDimRef.axis.x[0], boxDimRef.axis.x[0], boxDimRef.axis.x[0],0,

		calcEnergy.CellSize[0], calcEnergy.NumberOfCells[0], 
		calcEnergy.dev_CountAtomsInCell0,
		calcEnergy.AtomsInCells0, step);

	hipDeviceSynchronize();





#if ENSEMBLE == GEMC
	if ((calcEnergy.AtomCount[1]) < MAXTHREADSPERBLOCK)
		ThreadsPerBlock = calcEnergy.AtomCount[1];
	else
		ThreadsPerBlock = MAXTHREADSPERBLOCK;

	BlocksPerGrid = (calcEnergy.AtomCount[1] + ThreadsPerBlock - 1)/ ThreadsPerBlock;




	InitializeParticlePositions<<<BlocksPerGrid, ThreadsPerBlock>>>(
		calcEnergy.Gpu_x,
		calcEnergy.Gpu_y,
		calcEnergy.Gpu_z,
		calcEnergy.AtomCount[1],
		boxDimRef.axis.x[1], boxDimRef.axis.x[1], boxDimRef.axis.x[1],calcEnergy.AtomCount[0],

		calcEnergy.CellSize[1], calcEnergy.NumberOfCells[1], 
		calcEnergy.dev_CountAtomsInCell1,
		calcEnergy.AtomsInCells1,step);

	hipDeviceSynchronize();
#endif

}

#if ENSEMBLE == GEMC
void System::LoadAtomsToCellsVolumeMove(uint majKind)
{
	int ThreadsPerBlock, BlocksPerGrid;


	if ((calcEnergy.AtomCount[0]) < MAXTHREADSPERBLOCK)
		ThreadsPerBlock = calcEnergy.AtomCount[0];
	else
		ThreadsPerBlock = MAXTHREADSPERBLOCK;

	BlocksPerGrid = (calcEnergy.AtomCount[0] + ThreadsPerBlock - 1)/ ThreadsPerBlock;




	InitializeParticlePositions<<<BlocksPerGrid, ThreadsPerBlock>>>(
		calcEnergy.newX,
		calcEnergy.newY,
		calcEnergy.newZ,
		calcEnergy.AtomCount[0],
		((VolumeTransfer*)moves[majKind])->newDim.axis.x[0] , ((VolumeTransfer*)moves[majKind])->newDim.axis.y[0] , ((VolumeTransfer*)moves[majKind])->newDim.axis.z[0] ,0,

		calcEnergy.CellSize[0], calcEnergy.NumberOfCells[0], 
		calcEnergy.dev_CountAtomsInCell0,
		calcEnergy.AtomsInCells0,step);

	hipDeviceSynchronize();


	if ((calcEnergy.AtomCount[1]) < MAXTHREADSPERBLOCK)
		ThreadsPerBlock = calcEnergy.AtomCount[1];
	else
		ThreadsPerBlock = MAXTHREADSPERBLOCK;

	BlocksPerGrid = (calcEnergy.AtomCount[1] + ThreadsPerBlock - 1)/ ThreadsPerBlock;

	InitializeParticlePositions<<<BlocksPerGrid, ThreadsPerBlock>>>(
		calcEnergy.newX,
		calcEnergy.newY,
		calcEnergy.newZ,
		calcEnergy.AtomCount[1],
		((VolumeTransfer*)moves[majKind])->newDim.axis.x[1] , ((VolumeTransfer*)moves[majKind])->newDim.axis.y[1] , ((VolumeTransfer*)moves[majKind])->newDim.axis.z[1],calcEnergy.AtomCount[0],

		calcEnergy.CellSize[1], calcEnergy.NumberOfCells[1], 
		calcEnergy.dev_CountAtomsInCell1,
		calcEnergy.AtomsInCells1,step);

	hipDeviceSynchronize();


}
#endif



void System::DeviceQuery(){
	printf("================================================================\n\n");
	//Get device info
	hipDeviceProp_t prop;
	int count, driverVersion;
	size_t heapsize;

	hipGetDeviceCount(&count);
	if (count < 1) {
		printf("Error: No GPUs found.\nSimulation will Terminate\n");
		exit(3); 
	} //end if

	printf(" --- GPU System Configuration ---\n");
	hipDriverGetVersion(&driverVersion);
	printf("CUDA Version: %d.%d\n\n", driverVersion/1000, (driverVersion%100)/10);





	bool hasRequiredComputeCapability = false;

	for (int i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);
		hipDeviceGetLimit(&heapsize, hipLimitMallocHeapSize);
		printf(" --- General Information for device %d ---\n", i);
		printf("Name: %s\n", prop.name);
		printf("Compute capability: %d.%d\n", prop.major, prop.minor);
		printf("Clock rate: %.2f GHz\n", prop.clockRate * 1e-6f);
		printf(" --- Memory information for device %d ---\n", i);
		printf("Total global mem: %.2f GBytes\n", prop.totalGlobalMem/(1048576.0f*1024.0f));
		printf("Total constant mem: %.2f KBytes\n", prop.totalConstMem/(1024.0f));
		printf("Total dynamic heap mem: %.2f MBytes\n", heapsize/(1024.0f*1024.0f));
		printf(" --- Information on cores for device %d ---\n", i);
		printf("Streaming Multiprocessor (SM) count: %d\n", prop.multiProcessorCount);
		printf("Number of cores per SM: %d\n", _ConvertSMVer2Cores(prop.major, prop.minor));
		printf("Total number of cores: %d\n", _ConvertSMVer2Cores(prop.major, prop.minor) * prop.multiProcessorCount);
		printf("Shared mem per SM: %.2f KBytes\n", prop.sharedMemPerBlock/1024.0f);
		printf("Registers per SM: %d\n", prop.regsPerBlock);
		printf("Threads in warp: %d\n", prop.warpSize);
		printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("<<<<<<<=>>>>>>>\n\n");

		if (prop.major>= Min_CC_Major && prop.minor >= Min_CC_Minor)
		{
			hasRequiredComputeCapability=true;
		}
	}



	printf("================================================================\n\n");

}

SystemPotential System::ConvCellListSystemTotalEnergy()
{
	CreateAdjCellList();


	LoadAtomsToCells();

	SystemPotential pot= calcEnergy.SystemTotal();

	return pot;



}

#if ENSEMBLE == GEMC
SystemPotential System::NewConvCellListSystemTotalEnergy(uint majKind,SystemPotential curpot)
{

	CreateAdjCellListForScaledMols(majKind);

	LoadAtomsToCellsVolumeMove(majKind);

	SystemPotential pot= calcEnergy.CalculateNewEnergyCellList(((VolumeTransfer*)moves[majKind])->newDim,curpot, step);

	return pot;



}

SystemPotential System::NewConvCellListSystemTotalEnergyOneBox(uint majKind, int bPick)
{

	CreateAdjCellListForScaledMols(majKind);

	LoadAtomsToCellsVolumeMove(majKind);

	SystemPotential pot= calcEnergy.CalculateNewEnergyCellListOneBox(((VolumeTransfer*)moves[majKind])->newDim, step,bPick);

	return pot;



}



#endif

void System::Init(Setup const& set)
{
	prng.Init(set.prng.prngMaker.prng);
#ifdef VARIABLE_VOLUME
	boxDimensions.Init(set.config.in.restart, 
		set.config.sys.volume, set.pdb.cryst,
		statV.forcefield.rCut,
		statV.forcefield.rCutSq);
#endif
#ifdef VARIABLE_PARTICLE_NUMBER
	molLookup.Init(statV.mol, set.pdb.atoms); 
#endif
	moveSettings.Init(statV);
	//Note... the following calls use box iterators, so must come after
	//the molecule lookup initialization, in case we're in a constant 
	//particle/molecule ensemble, e.g. NVT
	coordinates.InitFromPDB(set.pdb.atoms);
	com.CalcCOM();

	DeviceQuery();

	LoadDataToGPU();

	// micro cell list
	LoadMolsToCells();

	#ifdef CELL_LIST
	potential =  ConvCellListSystemTotalEnergy();
#else

	potential = calcEnergy.SystemTotal();
#endif



#if ENSEMBLE == GEMC
	hipMemcpy(calcEnergy.newX, calcEnergy.Gpu_x, sizeof(double) * calcEnergy.currentCoords.Count(), hipMemcpyDeviceToDevice);
	hipMemcpy(calcEnergy.newY, calcEnergy.Gpu_y, sizeof(double) * calcEnergy.currentCoords.Count(), hipMemcpyDeviceToDevice);
	hipMemcpy(calcEnergy.newZ, calcEnergy.Gpu_z, sizeof(double) * calcEnergy.currentCoords.Count(), hipMemcpyDeviceToDevice);
	hipMemcpy(calcEnergy.newCOMX, calcEnergy.Gpu_COMX, sizeof(double) * calcEnergy.currentCOM.Count(), hipMemcpyDeviceToDevice);
	hipMemcpy(calcEnergy.newCOMY, calcEnergy.Gpu_COMY, sizeof(double) * calcEnergy.currentCOM.Count(), hipMemcpyDeviceToDevice);
	hipMemcpy(calcEnergy.newCOMZ, calcEnergy.Gpu_COMZ, sizeof(double) * calcEnergy.currentCOM.Count(), hipMemcpyDeviceToDevice);

#endif

	hipMemcpy(calcEnergy.Gpu_Potential, &potential, sizeof(SystemPotential), hipMemcpyHostToDevice );

	InitMoves();
}

void System::InitMoves()
{
	moves[mv::DISPLACE] = new Translate(*this, statV);
	moves[mv::ROTATE] = new Rotate(*this, statV);
#if ENSEMBLE == GEMC
	moves[mv::VOL_TRANSFER] = new VolumeTransfer(*this, statV);
#endif
#if ENSEMBLE == GEMC || ENSEMBLE == GCMC
	moves[mv::MOL_TRANSFER] = new MoleculeTransfer(*this, statV);
#endif
}

void System::ChooseAndRunMove(const uint step)
{
	double draw=0;
	uint majKind=0;
	PickMove(majKind, draw);
	RunMove(majKind, draw, step);
}
void System::PickMove(uint & kind, double & draw)
{ 
	prng.PickArbDist(kind, draw, statV.movePerc, statV.totalPerc, 
		mv::MOVE_KINDS_TOTAL);
}

void System::RunMove(uint majKind, double draw,const uint step)
{ 


#if ENSEMBLE == GCMC
	const int diff=1;
#else
	const int diff=0;
#endif

	////return now if move targets molecule and there's none in that box.
	uint rejectState = SetParams(majKind, draw);

	//If single atom, redo move as displacement
	if (rejectState == mv::fail_state::ROTATE_ON_SINGLE_ATOM) {
		majKind = mv::DISPLACE;
		Translate * disp = static_cast<Translate *>(moves[mv::DISPLACE]);
		Rotate * rot = static_cast<Rotate *>(moves[mv::ROTATE]);
		rejectState = disp->ReplaceRot(*rot);
	}

	switch (majKind) {
	case 0:


#ifdef CELL_LIST	
		RunDisplaceMoveUsingCellList(rejectState, majKind);
#else
		RunDisplaceMove(rejectState, majKind);
#endif
		break;

	case 1:


#ifdef CELL_LIST
		RunRotateMoveUsingCellList(rejectState, majKind);
#else
		RunRotateMove(rejectState, majKind);
#endif

		break;
#if ENSEMBLE == GEMC
	case 2:

#ifdef CELL_LIST
		RunVolumeMoveCell(rejectState, majKind,  this);
#else
		RunVolumeMove(rejectState, majKind,this);
#endif

		break;
#endif

#if ENSEMBLE == GEMC || ENSEMBLE == GCMC

	case 3-diff:

		RunMolTransferMove(rejectState, majKind, this);
		break;
#endif
	}


}
uint System::SetParams(const uint kind, const double draw) 
{ return moves[kind]->Prep(draw, statV.movePerc[kind]); }

uint System::Transform(const uint kind) { return moves[kind]->Transform(); }

void System::CalcEn(const uint kind) { moves[kind]->CalcEn(); }

void System::Accept(const uint kind, const uint rejectState, const uint step)
{ moves[kind]->Accept(rejectState,step); }

/////////////////////////////////////////

// GPU Code

// Author: Kamel Rushaidat

////////////////////////////////////////

// celllist methods for moves


void System::RunDisplaceMoveUsingCellList(uint rejectState, uint majKind)

{
	if (rejectState == mv::fail_state::NO_FAIL)
	{ rejectState = Transform(majKind); }

	bool resultFromMove[1];
	resultFromMove[0] = false;

	if (rejectState == mv::fail_state::NO_FAIL ) {

		int selectedBox = ((Translate*)moves[majKind])->Getb();
		int Boxoffset;
		int cellOffset;
		int cellrangeOffset;

		if (selectedBox == 0)
		{ Boxoffset = 0; 


		cellOffset=0;
		cellrangeOffset=0;



		}
		else
		{ Boxoffset = calcEnergy.MolCount[0];


		cellOffset= calcEnergy.TotalCellsPerBox[0];
		cellrangeOffset= calcEnergy.TotalCellsPerBox[0]*MAX_ATOMS_PER_CELL;
		}


		double max = moveSettings.Scale(mv::GetMoveSubIndex(mv::DISPLACE, selectedBox));
		XYZ shift = prng.SymXYZ(max);


		int len;
		len = calcEnergy.mols.kinds[ ((Translate*)moves[majKind])->Getmk()].numAtoms;
		int kindStart = molLookupRef.boxAndKindStart[selectedBox * molLookupRef.numKinds + ((Translate*)moves[majKind])->Getmk()];

		MTRand *r = prng.gen;
		double randToSend = (*r)();



		if (selectedBox < BOXES_WITH_U_NB)
		{

			TryTransformGpuCellList <<<  2* len,calcEnergy.BlockSize>>>(
				calcEnergy.tempCoordsX, 
				calcEnergy.tempCoordsY,
				calcEnergy.tempCoordsZ,
				calcEnergy.NoOfAtomsPerMol,
				calcEnergy.Gpu_atomKinds,
				calcEnergy.Gpu_Potential,
				calcEnergy.Gpu_x,
				calcEnergy.Gpu_y,
				calcEnergy.Gpu_z,
				calcEnergy.Gpu_COMX, 
				calcEnergy.Gpu_COMY,
				calcEnergy.Gpu_COMZ,
				shift,
				boxDimRef.axis.x[selectedBox],
				boxDimRef.axis.y[selectedBox],
				boxDimRef.axis.z[selectedBox],
				calcEnergy.EdgeAdjust[selectedBox* 3],
				calcEnergy.EdgeAdjust[selectedBox* 3+1],
				calcEnergy.EdgeAdjust[selectedBox* 3+2],
				calcEnergy.CellDim[selectedBox* 3],
				calcEnergy.CellDim[selectedBox* 3+1],
				calcEnergy.CellDim[selectedBox* 3+2],
				calcEnergy.CellsPerDim[selectedBox* 3],
				calcEnergy.CellsPerDim[selectedBox* 3+1],
				calcEnergy.CellsPerDim[selectedBox* 3+2],
				cellOffset,
				cellrangeOffset,
				calcEnergy.CellDim[selectedBox* 3]*calcEnergy.CellDim[selectedBox* 3+1]*calcEnergy.CellDim[selectedBox* 3+ 2],
				calcEnergy.atomCountrs,
				calcEnergy.atomCells,
				calcEnergy.TotalCellsPerBox[selectedBox], 
				calcEnergy.atomsMoleculeNo,
				calcEnergy.Gpu_kIndex,
				calcEnergy.Gpu_sigmaSq,
				calcEnergy.Gpu_epsilon_cn,
				calcEnergy.Gpu_nOver6,
				calcEnergy.Gpu_epsilon_cn_6,
				moves[majKind]->beta,
				randToSend,
				calcEnergy.Gpu_start,
				len,
				boxDimRef.halfAx.x[selectedBox],
				boxDimRef.halfAx.y[selectedBox],
				boxDimRef.halfAx.z[selectedBox],
				Boxoffset,
				calcEnergy.MolCount[selectedBox],
				((Translate*)moves[majKind])->GetmOff() + kindStart,
				calcEnergy.forcefield.particles->NumKinds(),
				boxDimRef.rCut,
				((Translate*)moves[majKind])->Getmk(),
				boxDimRef.rCutSq,
				calcEnergy.dev_EnergyContrib,
				calcEnergy.dev_VirialContrib,
				selectedBox,
				calcEnergy.Gpu_result,
				calcEnergy.Gpu_partn
				);
			hipMemcpy(resultFromMove, calcEnergy.Gpu_result, sizeof(bool), hipMemcpyDeviceToHost);

			if (resultFromMove[0]==1)
				hipMemcpy(&potential, calcEnergy.Gpu_Potential, sizeof(SystemPotential), hipMemcpyDeviceToHost);// remove ? 



		}


	}
	hipError_t  code = hipGetLastError();
	if (code != hipSuccess) {
		printf ("Cuda error at end of energy calc of Displace run at step %d-- %s\n", step, hipGetErrorString(code));
		exit(2);
	}


	((Translate*)moves[majKind])->AcceptGPU(rejectState, resultFromMove[0] ,step);

	hipDeviceSynchronize();


	if (code != hipSuccess) {
		printf ("Cuda error at Displace run at step %d-- %s\n", step, hipGetErrorString(code));
		exit(2);
	}


}

void System::RunRotateMoveUsingCellList(uint rejectState, uint majKind)

{
	if (rejectState == mv::fail_state::NO_FAIL)
	{ rejectState = Transform(majKind); }

	bool resultFromMove[1];
	resultFromMove[0] = false;

	if (rejectState == mv::fail_state::NO_FAIL ) {

		int selectedBox = ((Rotate*)moves[majKind])->Getb();
		int Boxoffset;
		int cellOffset;
		int cellrangeOffset;

		if (selectedBox == 0)
		{ Boxoffset = 0; 


		cellOffset=0;
		cellrangeOffset=0;



		}
		else
		{ Boxoffset = calcEnergy.MolCount[0];


		cellOffset= calcEnergy.TotalCellsPerBox[0];
		cellrangeOffset= calcEnergy.TotalCellsPerBox[0]*MAX_ATOMS_PER_CELL;
		}


		double max = moveSettings.Scale(mv::GetMoveSubIndex(mv::ROTATE, selectedBox));

		double sym=prng.Sym(max);

		XYZ PickonSphere= prng.PickOnUnitSphere();

		RotationMatrix matrix = RotationMatrix::FromAxisAngle(sym,PickonSphere );



		int len;
		len = calcEnergy.mols.kinds[ ((Rotate*)moves[majKind])->Getmk()].numAtoms;
		int kindStart = molLookupRef.boxAndKindStart[selectedBox * molLookupRef.numKinds + ((Rotate*)moves[majKind])->Getmk()];


		MTRand *r = prng.gen;
		double randToSend = (*r)();


		if (selectedBox < BOXES_WITH_U_NB)
		{

			TryRotateGpuCellList <<<2* len,calcEnergy.BlockSize>>>(
				calcEnergy.tempCoordsX, 
				calcEnergy.tempCoordsY,
				calcEnergy.tempCoordsZ,
				calcEnergy.NoOfAtomsPerMol,
				calcEnergy.Gpu_atomKinds,
				calcEnergy.Gpu_Potential,
				calcEnergy.Gpu_x,
				calcEnergy.Gpu_y,
				calcEnergy.Gpu_z,
				calcEnergy.Gpu_COMX, 
				calcEnergy.Gpu_COMY,
				calcEnergy.Gpu_COMZ,
				matrix,
				boxDimRef.axis.x[selectedBox],
				boxDimRef.axis.y[selectedBox],
				boxDimRef.axis.z[selectedBox],
				calcEnergy.EdgeAdjust[selectedBox* 3],
				calcEnergy.EdgeAdjust[selectedBox* 3+1],
				calcEnergy.EdgeAdjust[selectedBox* 3+2],
				calcEnergy.CellDim[selectedBox* 3],
				calcEnergy.CellDim[selectedBox* 3+1],
				calcEnergy.CellDim[selectedBox* 3+2],
				calcEnergy.CellsPerDim[selectedBox* 3],
				calcEnergy.CellsPerDim[selectedBox* 3+1],
				calcEnergy.CellsPerDim[selectedBox* 3+2],
				cellOffset,
				cellrangeOffset,
				calcEnergy.CellDim[selectedBox* 3]*calcEnergy.CellDim[selectedBox* 3+1]*calcEnergy.CellDim[selectedBox* 3+ 2],
				calcEnergy.atomCountrs,
				calcEnergy.atomCells,
				calcEnergy.TotalCellsPerBox[selectedBox], 
				calcEnergy.atomsMoleculeNo,
				calcEnergy.Gpu_kIndex,
				calcEnergy.Gpu_sigmaSq,
				calcEnergy.Gpu_epsilon_cn,
				calcEnergy.Gpu_nOver6,
				calcEnergy.Gpu_epsilon_cn_6,
				moves[majKind]->beta,
				randToSend,
				calcEnergy.Gpu_start,
				len,
				boxDimRef.halfAx.x[selectedBox],
				boxDimRef.halfAx.y[selectedBox],
				boxDimRef.halfAx.z[selectedBox],
				Boxoffset,
				calcEnergy.MolCount[selectedBox],
				((Rotate*)moves[majKind])->GetmOff() + kindStart,
				calcEnergy.forcefield.particles->NumKinds(),
				boxDimRef.rCut,
				((Rotate*)moves[majKind])->Getmk(),
				boxDimRef.rCutSq,
				calcEnergy.dev_EnergyContrib,
				calcEnergy.dev_VirialContrib,
				selectedBox,
				calcEnergy.Gpu_result,
				calcEnergy.Gpu_partn
				);
			hipMemcpy(resultFromMove, calcEnergy.Gpu_result, sizeof(bool), hipMemcpyDeviceToHost);
			if (resultFromMove[0]==1)
				hipMemcpy(&potential, calcEnergy.Gpu_Potential, sizeof(SystemPotential), hipMemcpyDeviceToHost);
		}


	}


	((Rotate*)moves[majKind])->AcceptGPU(rejectState, resultFromMove[0] ,step);

	hipDeviceSynchronize();
	hipError_t  code = hipGetLastError();

	if (code != hipSuccess) {
		printf ("Cuda error at Displace run-- %s\n", hipGetErrorString(code));
		exit(2);
	}
}


void System::RunDisplaceMove(uint rejectState, uint majKind)

{
	if (rejectState == mv::fail_state::NO_FAIL)
	{ rejectState = Transform(majKind); }

	bool resultFromMove[1];
	resultFromMove[0] = false;

	if (rejectState == mv::fail_state::NO_FAIL ) {

		int selectedBox = ((Translate*)moves[majKind])->Getb();
		int offset;

		if (selectedBox == 0)
		{ offset = 0; }
		else
		{ offset = calcEnergy.MolCount[0]; }

		int ThreadsPerBlock1 = 0;
		int BlocksPerGrid1 = 0;

		ThreadsPerBlock1 = MAXTHREADSPERBLOCK;

		if(ThreadsPerBlock1 == 0)
		{ ThreadsPerBlock1 = 1; }

		BlocksPerGrid1 = ((calcEnergy.MolCount[selectedBox]) + ThreadsPerBlock1 - 1) / ThreadsPerBlock1;

		if (BlocksPerGrid1 == 0)
		{ BlocksPerGrid1 = 1; }

		double * dev_EnergyContrib, * dev_VirialContrib;
		hipMalloc((void**) &dev_EnergyContrib, 4 * BlocksPerGrid1 * sizeof(double));
		hipMalloc((void**) &dev_VirialContrib, 4 * BlocksPerGrid1 * sizeof(double));


		double max = moveSettings.Scale(mv::GetMoveSubIndex(mv::DISPLACE, selectedBox));
		XYZ shift = prng.SymXYZ(max);


		int len;
		len = calcEnergy.mols.kinds[ ((Translate*)moves[majKind])->Getmk()].numAtoms;
		int kindStart = molLookupRef.boxAndKindStart[selectedBox * molLookupRef.numKinds + ((Translate*)moves[majKind])->Getmk()];



		MTRand *r = prng.gen;
		double randToSend = (*r)();

		//Intermolecular result;
		if (selectedBox < BOXES_WITH_U_NB)
		{

			TryTransformGpu <<< BlocksPerGrid1, ThreadsPerBlock1, len*3*sizeof(double)>>>(
				calcEnergy.NoOfAtomsPerMol, calcEnergy.Gpu_atomKinds, calcEnergy.Gpu_Potential,
				calcEnergy.Gpu_x,  calcEnergy.Gpu_y,  calcEnergy.Gpu_z,
				calcEnergy.Gpu_COMX,  calcEnergy.Gpu_COMY, calcEnergy.Gpu_COMZ,
				shift, boxDimRef.axis.x[selectedBox], boxDimRef.axis.y[selectedBox],  boxDimRef.axis.z[selectedBox],
				calcEnergy.Gpu_kIndex,
				calcEnergy.Gpu_sigmaSq,
				calcEnergy.Gpu_epsilon_cn,
				calcEnergy.Gpu_nOver6,
				calcEnergy.Gpu_epsilon_cn_6,
				moves[majKind]->beta,
				randToSend,
				calcEnergy.Gpu_start,
				len,
				boxDimRef.halfAx.x[selectedBox],
				boxDimRef.halfAx.y[selectedBox],
				boxDimRef.halfAx.z[selectedBox],
				offset,
				calcEnergy.MolCount[selectedBox],
				((Translate*)moves[majKind])->GetmOff() + kindStart,
				calcEnergy.forcefield.particles->NumKinds(),
				boxDimRef.rCut,
				((Translate*)moves[majKind])->Getmk(),
				boxDimRef.rCutSq,
				dev_EnergyContrib,
				dev_VirialContrib,
				selectedBox,
				calcEnergy.Gpu_result,
				calcEnergy.Gpu_partn
				);
			hipMemcpy(resultFromMove, calcEnergy.Gpu_result, sizeof(bool), hipMemcpyDeviceToHost);
			hipMemcpy(&potential, calcEnergy.Gpu_Potential, sizeof(SystemPotential), hipMemcpyDeviceToHost);

		}



		hipFree (dev_EnergyContrib);
		hipFree(dev_VirialContrib);
	}

	((Translate*)moves[majKind])->AcceptGPU(rejectState, resultFromMove[0],step );
	hipDeviceSynchronize();
	hipError_t  code = hipGetLastError();

	if (code != hipSuccess) {
		printf ("Cuda error at Displace run-- %s\n", hipGetErrorString(code));
		exit(2);
	}


}
void System::RunRotateMove(uint rejectState, uint majKind )

{
	if (rejectState == mv::fail_state::NO_FAIL)
	{ rejectState = Transform(majKind); }

	bool resultFromMove[1];
	resultFromMove[0] = false;

	if (rejectState == mv::fail_state::NO_FAIL ) {

		int selectedBox = ((Rotate*)moves[majKind])->Getb();




		int offset;

		if (selectedBox == 0)
		{ offset = 0; }
		else
		{ offset = calcEnergy.MolCount[0]; }

		int ThreadsPerBlock1 = 0;
		int BlocksPerGrid1 = 0;

		ThreadsPerBlock1 = MAXTHREADSPERBLOCK;

		if(ThreadsPerBlock1 == 0)
		{ ThreadsPerBlock1 = 1; }

		BlocksPerGrid1 = ((calcEnergy.MolCount[selectedBox]) + ThreadsPerBlock1 - 1) / ThreadsPerBlock1;

		if (BlocksPerGrid1 == 0)
		{ BlocksPerGrid1 = 1; }

		double * dev_EnergyContrib, * dev_VirialContrib;
		hipMalloc((void**) &dev_EnergyContrib, 4 * BlocksPerGrid1 * sizeof(double));
		hipMalloc((void**) &dev_VirialContrib, 4 * BlocksPerGrid1 * sizeof(double));

		int molLen = calcEnergy.mols.kinds[ ((Rotate*)moves[majKind])->Getmk()].numAtoms;

		double max = moveSettings.Scale(mv::GetMoveSubIndex(mv::ROTATE, selectedBox));

		double sym=prng.Sym(max);

		XYZ PickonSphere= prng.PickOnUnitSphere();

		RotationMatrix matrix = RotationMatrix::FromAxisAngle(sym,PickonSphere );

		int kindStart = molLookupRef.boxAndKindStart[selectedBox * molLookupRef.numKinds + ((Translate*)moves[majKind])->Getmk()];
		MTRand *r = prng.gen;
		double randToSend = (*r)();


		if (selectedBox < BOXES_WITH_U_NB)
		{

			TryRotateGpu <<< BlocksPerGrid1, ThreadsPerBlock1, molLen* sizeof(double)*3>>>(
				calcEnergy.NoOfAtomsPerMol,
				calcEnergy.Gpu_atomKinds,
				calcEnergy.Gpu_Potential,
				matrix,
				calcEnergy.Gpu_x,
				calcEnergy.Gpu_y,
				calcEnergy.Gpu_z,
				calcEnergy.Gpu_COMX,
				calcEnergy.Gpu_COMY,
				calcEnergy.Gpu_COMZ,
				boxDimRef.axis.x[selectedBox],
				boxDimRef.axis.y[selectedBox],
				boxDimRef.axis.z[selectedBox],
				calcEnergy.Gpu_kIndex,
				calcEnergy.Gpu_sigmaSq,
				calcEnergy. Gpu_epsilon_cn,
				calcEnergy.Gpu_nOver6,
				calcEnergy.Gpu_epsilon_cn_6,
				moves[majKind]->beta,
				randToSend ,
				calcEnergy.Gpu_start,
				molLen,
				boxDimRef.halfAx.x[selectedBox],
				boxDimRef.halfAx.x[selectedBox],
				boxDimRef.halfAx.x[selectedBox],
				offset,
				calcEnergy.MolCount[selectedBox],
				((Rotate*)moves[majKind])->GetmOff() + kindStart,
				calcEnergy.forcefield.particles->NumKinds(),
				boxDimRef.rCut,
				((Rotate*)moves[majKind])->Getmk(),
				boxDimRef.rCutSq,
				dev_EnergyContrib,
				dev_VirialContrib,
				selectedBox,
				calcEnergy.Gpu_result,
				calcEnergy.Gpu_partn
				);
			hipMemcpy(resultFromMove, calcEnergy.Gpu_result, sizeof(bool), hipMemcpyDeviceToHost);
			hipMemcpy(&potential, calcEnergy.Gpu_Potential, sizeof(SystemPotential), hipMemcpyDeviceToHost);
		}

		hipFree (dev_EnergyContrib);
		hipFree(dev_VirialContrib);
	}

	((Rotate*)moves[majKind])->AcceptGPU(rejectState, resultFromMove[0],step  );
	hipDeviceSynchronize();
	hipError_t  code = hipGetLastError();

	if (code != hipSuccess) {
		printf ("Cuda error at Rotate Move-- %s\n", hipGetErrorString(code));
		exit(2);
	}
}

#if ENSEMBLE == GEMC
void System::RunVolumeMove(uint rejectState, uint majKind,System  * sys)

{
	if(((VolumeTransfer*)moves[majKind])->GEMC_KIND == mv::GEMC_NVT)
	{
		int srcBox, distBox;

		if (rejectState == mv::fail_state::NO_FAIL) {
			rejectState = Transform(majKind);
			int ThreadsPerBlock1 = 0;
			int BlocksPerGrid1 = 0;
			double scaleO, scaleN;
			double randN;
			srcBox =0;
			distBox = 1;


			scaleO = ((VolumeTransfer*)moves[majKind])->scaleO;
			scaleN = ((VolumeTransfer*)moves[majKind])->scaleN;

			randN = ((VolumeTransfer*)moves[majKind])->randN;

			if (rejectState == mv::fail_state::NO_FAIL) {
				if (calcEnergy.MolCount[srcBox] < MAXTHREADSPERBLOCK)
				{ ThreadsPerBlock1 = calcEnergy.MolCount[srcBox]; }
				else
				{ ThreadsPerBlock1 = MAXTHREADSPERBLOCK; }

				if(ThreadsPerBlock1 == 0)
				{ ThreadsPerBlock1 = 1; }

				BlocksPerGrid1 = ((calcEnergy.MolCount[srcBox]) + ThreadsPerBlock1 - 1) / ThreadsPerBlock1;

				if (BlocksPerGrid1 == 0)
				{ BlocksPerGrid1 = 1; }

				ScaleMolecules <<< BlocksPerGrid1, ThreadsPerBlock1, 0, calcEnergy.stream0>>>(calcEnergy.NoOfAtomsPerMol,
					calcEnergy.Gpu_kIndex, calcEnergy.Gpu_x, calcEnergy.Gpu_y,  calcEnergy.Gpu_z,
					calcEnergy.Gpu_COMX,  calcEnergy.Gpu_COMY,  calcEnergy.Gpu_COMZ,
					calcEnergy.newX, calcEnergy.newY, calcEnergy.newZ,
					calcEnergy.newCOMX, calcEnergy.newCOMY, calcEnergy.newCOMZ,
					scaleO, calcEnergy.MolCount[srcBox],
					((VolumeTransfer*)moves[majKind])->newDim.axis.x[srcBox],
					((VolumeTransfer*)moves[majKind])->newDim.axis.y[srcBox],
					((VolumeTransfer*)moves[majKind])->newDim.axis.z[srcBox],
					boxDimRef.axis.x[srcBox],
					boxDimRef.axis.y[srcBox],
					boxDimRef.axis.z[srcBox],
					boxDimRef.halfAx.x[srcBox],
					boxDimRef.halfAx.x[srcBox],
					boxDimRef.halfAx.x[srcBox],
					(srcBox == 0) ? 0 : calcEnergy.MolCount[0],
					calcEnergy.Gpu_start
					);
				hipStreamSynchronize(calcEnergy.stream0);

				if (calcEnergy.MolCount[distBox] < MAXTHREADSPERBLOCK)
					ThreadsPerBlock1 = calcEnergy.MolCount[distBox];
				else
				{ ThreadsPerBlock1 = MAXTHREADSPERBLOCK; }
				if(ThreadsPerBlock1 == 0)
					ThreadsPerBlock1 = 1;
				BlocksPerGrid1 = ((calcEnergy.MolCount[distBox]) + ThreadsPerBlock1 - 1) / ThreadsPerBlock1; 
				if (BlocksPerGrid1 == 0) BlocksPerGrid1 = 1;
				ScaleMolecules <<< BlocksPerGrid1, ThreadsPerBlock1, 0, calcEnergy.stream1>>>(calcEnergy.NoOfAtomsPerMol,
					calcEnergy.Gpu_kIndex, calcEnergy.Gpu_x, calcEnergy.Gpu_y,  calcEnergy.Gpu_z,
					calcEnergy.Gpu_COMX,  calcEnergy.Gpu_COMY,  calcEnergy.Gpu_COMZ,
					calcEnergy.newX, calcEnergy.newY, calcEnergy.newZ,
					calcEnergy.newCOMX, calcEnergy.newCOMY, calcEnergy.newCOMZ,
					scaleN, calcEnergy.MolCount[distBox],
					((VolumeTransfer*)moves[majKind])->newDim.axis.x[distBox],
					((VolumeTransfer*)moves[majKind])->newDim.axis.y[distBox],
					((VolumeTransfer*)moves[majKind])->newDim.axis.z[distBox],
					boxDimRef.axis.x[distBox],
					boxDimRef.axis.y[distBox],
					boxDimRef.axis.z[distBox],
					boxDimRef.halfAx.x[distBox],
					boxDimRef.halfAx.x[distBox],
					boxDimRef.halfAx.x[distBox],
					(distBox == 0) ? 0 : calcEnergy.MolCount[0],
					calcEnergy.Gpu_start
					);
				hipStreamSynchronize(calcEnergy.stream1);
			}
		}

		SystemPotential curpot ;
		SystemPotential newpot;

		if (rejectState == mv::fail_state::NO_FAIL ) {
			hipMemcpy(& curpot, calcEnergy.Gpu_Potential, sizeof(SystemPotential)  , hipMemcpyDeviceToHost);
			newpot =  calcEnergy.NewSystemInterGPU(step,  ((VolumeTransfer*)moves[majKind])->newDim, srcBox, distBox);


		}

		((VolumeTransfer*)moves[majKind])->AcceptGPU(rejectState, newpot, curpot, 0,step,sys);
		hipDeviceSynchronize();
		hipError_t code = hipGetLastError();

		if (code != hipSuccess) {
			printf ("Cuda error at volume move-- %s, LINE: %d\n", hipGetErrorString(code), __LINE__);
			exit(2);
		}
	}
	else
	{
		uint bPick;
		if (rejectState == mv::fail_state::NO_FAIL) {
			rejectState = Transform(majKind);
			int ThreadsPerBlock1 = 0;
			int BlocksPerGrid1 = 0;
			double scaleO, scaleN, scaleP;
			double randN;
			bPick  = ((VolumeTransfer*)moves[majKind])->bPick;
			scaleO = ((VolumeTransfer*)moves[majKind])->scaleO;
			scaleN = ((VolumeTransfer*)moves[majKind])->scaleN;
			scaleP = ((VolumeTransfer*)moves[majKind])->scaleP;
			randN  = ((VolumeTransfer*)moves[majKind])->randN;
			if (rejectState == mv::fail_state::NO_FAIL) {


				if (calcEnergy.MolCount[bPick] < MAXTHREADSPERBLOCK)
				{ ThreadsPerBlock1 = calcEnergy.MolCount[bPick]; }
				else
				{ ThreadsPerBlock1 = MAXTHREADSPERBLOCK; }

				if(ThreadsPerBlock1 == 0)
				{ ThreadsPerBlock1 = 1; }

				BlocksPerGrid1 = ((calcEnergy.MolCount[bPick]) + ThreadsPerBlock1 - 1) / ThreadsPerBlock1;

				if (BlocksPerGrid1 == 0)
				{ BlocksPerGrid1 = 1; }

				ScaleMolecules <<< BlocksPerGrid1, ThreadsPerBlock1, 0, calcEnergy.stream0>>>(calcEnergy.NoOfAtomsPerMol,
					calcEnergy.Gpu_kIndex, calcEnergy.Gpu_x, calcEnergy.Gpu_y,  calcEnergy.Gpu_z,
					calcEnergy.Gpu_COMX,  calcEnergy.Gpu_COMY,  calcEnergy.Gpu_COMZ,
					calcEnergy.newX, calcEnergy.newY, calcEnergy.newZ,
					calcEnergy.newCOMX, calcEnergy.newCOMY, calcEnergy.newCOMZ,
					scaleP, calcEnergy.MolCount[bPick],
					((VolumeTransfer*)moves[majKind])->newDim.axis.x[bPick],
					((VolumeTransfer*)moves[majKind])->newDim.axis.y[bPick],
					((VolumeTransfer*)moves[majKind])->newDim.axis.z[bPick],
					boxDimRef.axis.x[bPick],
					boxDimRef.axis.y[bPick],
					boxDimRef.axis.z[bPick],
					boxDimRef.halfAx.x[bPick],
					boxDimRef.halfAx.x[bPick],
					boxDimRef.halfAx.x[bPick],
					(bPick == 0) ? 0 : calcEnergy.MolCount[0],
					calcEnergy.Gpu_start
					);
				hipStreamSynchronize(calcEnergy.stream0);
			}
		}

		SystemPotential curpot ;
		SystemPotential newpot;

		if (rejectState == mv::fail_state::NO_FAIL ) {
			hipMemcpy(& curpot, calcEnergy.Gpu_Potential, sizeof(SystemPotential)  , hipMemcpyDeviceToHost);
			newpot =  calcEnergy.NewSystemInterGPUOneBox(  ((VolumeTransfer*)moves[majKind])->newDim, bPick);
		}

		((VolumeTransfer*)moves[majKind])->AcceptGPU(rejectState, newpot, curpot, bPick,step,sys);
		hipDeviceSynchronize();
		hipError_t code = hipGetLastError();

		if (code != hipSuccess) {
			printf ("Cuda error at volume move-- %s, LINE: %d\n", hipGetErrorString(code), __LINE__);
			exit(2);
		}
	}
}



// cell list volume move 

void System::RunVolumeMoveCell(uint rejectState, uint majKind, System * sys)

{
	if(((VolumeTransfer*)moves[majKind])->GEMC_KIND == mv::GEMC_NVT)
	{
		int srcBox, distBox;

		if (rejectState == mv::fail_state::NO_FAIL) {
			rejectState = Transform(majKind);
			int ThreadsPerBlock1 = 0;
			int BlocksPerGrid1 = 0;
			double scaleO, scaleN;
			double randN;
			srcBox = ((VolumeTransfer*)moves[majKind])->b_i;
			distBox = ((VolumeTransfer*)moves[majKind])->b_ii;
			scaleO = ((VolumeTransfer*)moves[majKind])->scaleO;
			scaleN = ((VolumeTransfer*)moves[majKind])->scaleN;
			randN = ((VolumeTransfer*)moves[majKind])->randN;

			if (rejectState == mv::fail_state::NO_FAIL) {
				if (calcEnergy.MolCount[srcBox] < MAXTHREADSPERBLOCK)
				{ ThreadsPerBlock1 = calcEnergy.MolCount[srcBox]; }
				else
				{ ThreadsPerBlock1 = MAXTHREADSPERBLOCK; }

				if(ThreadsPerBlock1 == 0)
				{ ThreadsPerBlock1 = 1; }

				BlocksPerGrid1 = ((calcEnergy.MolCount[srcBox]) + ThreadsPerBlock1 - 1) / ThreadsPerBlock1;

				if (BlocksPerGrid1 == 0)
				{ BlocksPerGrid1 = 1; }

				ScaleMolecules <<< BlocksPerGrid1, ThreadsPerBlock1, 0, calcEnergy.stream0>>>(calcEnergy.NoOfAtomsPerMol,
					calcEnergy.Gpu_kIndex, calcEnergy.Gpu_x, calcEnergy.Gpu_y,  calcEnergy.Gpu_z,
					calcEnergy.Gpu_COMX,  calcEnergy.Gpu_COMY,  calcEnergy.Gpu_COMZ,
					calcEnergy.newX, calcEnergy.newY, calcEnergy.newZ,
					calcEnergy.newCOMX, calcEnergy.newCOMY, calcEnergy.newCOMZ,
					scaleO, calcEnergy.MolCount[srcBox],
					((VolumeTransfer*)moves[majKind])->newDim.axis.x[srcBox],
					((VolumeTransfer*)moves[majKind])->newDim.axis.y[srcBox],
					((VolumeTransfer*)moves[majKind])->newDim.axis.z[srcBox],
					boxDimRef.axis.x[srcBox],
					boxDimRef.axis.y[srcBox],
					boxDimRef.axis.z[srcBox],
					boxDimRef.halfAx.x[srcBox],
					boxDimRef.halfAx.x[srcBox],
					boxDimRef.halfAx.x[srcBox],
					(srcBox == 0) ? 0 : calcEnergy.MolCount[0],
					calcEnergy.Gpu_start
					);
				hipStreamSynchronize(calcEnergy.stream0);

				if (calcEnergy.MolCount[distBox] < MAXTHREADSPERBLOCK)
					ThreadsPerBlock1 = calcEnergy.MolCount[distBox];
				else
				{ ThreadsPerBlock1 = MAXTHREADSPERBLOCK; }
				if(ThreadsPerBlock1 == 0)
					ThreadsPerBlock1 = 1;
				BlocksPerGrid1 = ((calcEnergy.MolCount[distBox]) + ThreadsPerBlock1 - 1) / ThreadsPerBlock1; 
				if (BlocksPerGrid1 == 0) BlocksPerGrid1 = 1;
				ScaleMolecules <<< BlocksPerGrid1, ThreadsPerBlock1, 0, calcEnergy.stream1>>>(calcEnergy.NoOfAtomsPerMol,
					calcEnergy.Gpu_kIndex, calcEnergy.Gpu_x, calcEnergy.Gpu_y,  calcEnergy.Gpu_z,
					calcEnergy.Gpu_COMX,  calcEnergy.Gpu_COMY,  calcEnergy.Gpu_COMZ,
					calcEnergy.newX, calcEnergy.newY, calcEnergy.newZ,
					calcEnergy.newCOMX, calcEnergy.newCOMY, calcEnergy.newCOMZ,
					scaleN, calcEnergy.MolCount[distBox],
					((VolumeTransfer*)moves[majKind])->newDim.axis.x[distBox],
					((VolumeTransfer*)moves[majKind])->newDim.axis.y[distBox],
					((VolumeTransfer*)moves[majKind])->newDim.axis.z[distBox],
					boxDimRef.axis.x[distBox],
					boxDimRef.axis.y[distBox],
					boxDimRef.axis.z[distBox],
					boxDimRef.halfAx.x[distBox],
					boxDimRef.halfAx.x[distBox],
					boxDimRef.halfAx.x[distBox],
					(distBox == 0) ? 0 : calcEnergy.MolCount[0],
					calcEnergy.Gpu_start
					);
				hipStreamSynchronize(calcEnergy.stream1);
			}
		}

		SystemPotential curpot ;
		SystemPotential newpot;

		if (rejectState == mv::fail_state::NO_FAIL ) {
			hipMemcpy(& curpot, calcEnergy.Gpu_Potential, sizeof(SystemPotential)  , hipMemcpyDeviceToHost);

			int numberOfCellsB0= (int) floor(((VolumeTransfer*)moves[majKind])->newDim.axis.x[0] / calcEnergy.currentAxes.rCut);// now for box 0, but update to do each box alone
			int numberOfCellsB1= (int) floor(((VolumeTransfer*)moves[majKind])->newDim.axis.x[1] / calcEnergy.currentAxes.rCut);// now for box 0, but update to do each box alone

			numberOfCellsB0 = numberOfCellsB0*numberOfCellsB0*numberOfCellsB0;
			numberOfCellsB1 = numberOfCellsB1*numberOfCellsB1*numberOfCellsB1;


			if (numberOfCellsB0>=27 && numberOfCellsB1>=27)
				newpot = NewConvCellListSystemTotalEnergy(majKind,curpot);
			else
				newpot =  calcEnergy.NewSystemInterGPU( step, ((VolumeTransfer*)moves[majKind])->newDim, srcBox, distBox);


		}

		((VolumeTransfer*)moves[majKind])->AcceptGPU(rejectState, newpot, curpot, 0,step, sys);
		hipDeviceSynchronize();
		hipError_t code = hipGetLastError();

		if (code != hipSuccess) {
			printf ("Cuda error at volume move-- %s, LINE: %d\n", hipGetErrorString(code), __LINE__);
			exit(2);
		}
	}

	else// NPT
	{
		uint bPick;
		if (rejectState == mv::fail_state::NO_FAIL) {
			rejectState = Transform(majKind);
			int ThreadsPerBlock1 = 0;
			int BlocksPerGrid1 = 0;
			double scaleO, scaleN, scaleP;
			double randN;
			bPick  = ((VolumeTransfer*)moves[majKind])->bPick;
			scaleO = ((VolumeTransfer*)moves[majKind])->scaleO;
			scaleN = ((VolumeTransfer*)moves[majKind])->scaleN;
			scaleP = ((VolumeTransfer*)moves[majKind])->scaleP;
			randN  = ((VolumeTransfer*)moves[majKind])->randN;
			if (rejectState == mv::fail_state::NO_FAIL) {


				if (calcEnergy.MolCount[bPick] < MAXTHREADSPERBLOCK)
				{ ThreadsPerBlock1 = calcEnergy.MolCount[bPick]; }
				else
				{ ThreadsPerBlock1 = MAXTHREADSPERBLOCK; }

				if(ThreadsPerBlock1 == 0)
				{ ThreadsPerBlock1 = 1; }

				BlocksPerGrid1 = ((calcEnergy.MolCount[bPick]) + ThreadsPerBlock1 - 1) / ThreadsPerBlock1;

				if (BlocksPerGrid1 == 0)
				{ BlocksPerGrid1 = 1; }

				ScaleMolecules <<< BlocksPerGrid1, ThreadsPerBlock1, 0, calcEnergy.stream0>>>(calcEnergy.NoOfAtomsPerMol,
					calcEnergy.Gpu_kIndex, calcEnergy.Gpu_x, calcEnergy.Gpu_y,  calcEnergy.Gpu_z,
					calcEnergy.Gpu_COMX,  calcEnergy.Gpu_COMY,  calcEnergy.Gpu_COMZ,
					calcEnergy.newX, calcEnergy.newY, calcEnergy.newZ,
					calcEnergy.newCOMX, calcEnergy.newCOMY, calcEnergy.newCOMZ,
					scaleP, calcEnergy.MolCount[bPick],
					((VolumeTransfer*)moves[majKind])->newDim.axis.x[bPick],
					((VolumeTransfer*)moves[majKind])->newDim.axis.y[bPick],
					((VolumeTransfer*)moves[majKind])->newDim.axis.z[bPick],
					boxDimRef.axis.x[bPick],
					boxDimRef.axis.y[bPick],
					boxDimRef.axis.z[bPick],
					boxDimRef.halfAx.x[bPick],
					boxDimRef.halfAx.x[bPick],
					boxDimRef.halfAx.x[bPick],
					(bPick == 0) ? 0 : calcEnergy.MolCount[0],
					calcEnergy.Gpu_start
					);
				hipStreamSynchronize(calcEnergy.stream0);
			}
		}

		SystemPotential curpot ;
		SystemPotential newpot;

		if (rejectState == mv::fail_state::NO_FAIL ) {
			hipMemcpy(& curpot, calcEnergy.Gpu_Potential, sizeof(SystemPotential)  , hipMemcpyDeviceToHost);


			newpot =  calcEnergy.NewSystemInterGPUOneBox(  ((VolumeTransfer*)moves[majKind])->newDim, bPick);



		}

		((VolumeTransfer*)moves[majKind])->AcceptGPU(rejectState, newpot, curpot, bPick,step,sys);
		hipDeviceSynchronize();
		hipError_t code = hipGetLastError();

		if (code != hipSuccess) {
			printf ("Cuda error at volume move-- %s, LINE: %d\n", hipGetErrorString(code), __LINE__);
			exit(2);
		}
	}
}

#endif



#if ENSEMBLE == GEMC || ENSEMBLE == GCMC
void System::RunMolTransferMove(uint rejectState, uint majKind, System * sys)

{
	if (rejectState == mv::fail_state::NO_FAIL )
	{ rejectState = Transform(majKind); }

	if (rejectState == mv::fail_state::NO_FAIL ) {
		CalcEn(majKind);
	}

	((MoleculeTransfer*)moves[majKind])->AcceptGPU(rejectState, step,sys);
	hipDeviceSynchronize();
	hipError_t  code = hipGetLastError();

	if (code != hipSuccess) {
		printf ("Cuda error at Molecule transfer Move -- %s\n", hipGetErrorString(code));
		exit(2);
	}
}

#endif


